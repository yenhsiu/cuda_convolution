#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 256
#define BLOCK_SIZE	16


__global__ static void Conv(const float* a, int lda, const float* b, int ldb, float* c, int ldc)
{
	extern __shared__ float data[];
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
    const int idx = bid * blockDim.x + tid;
	const int row = idx / ldc;
	const int column = idx % ldc;
	int i,  j;
	
	for(i = 0; i < ldb*ldb; i ++){
		data[i] = b[i];
	}
	__syncthreads();
    // printf("bid %d \n",bid); 
    if(row < ldc && column < ldc) 
	{
		float t = 0;
		for (j = 0;j<ldb*ldb;j++)
		{
			t += a[ lda*row + column + lda*(j/ldc) + j%ldc ] * data[j];	     
		}
        // printf("t %.0f ",t);
        // printf("c %d \t",bid * blockDim.x + tid);
        if(t>255){c[bid * blockDim.x + tid]=255;}
        else if(t<0){ c[bid * blockDim.x + tid]=0;}
        else{ c[idx] = t;}
		// printf(" %.0f ",c[row * ldc + column]);
	}
}


clock_t conv(const float* a, int lda, const float* b, int ldb, float* c, int ldc)
{
	float *ac, *bc, *cc;
	clock_t start, end;

	start = clock();

    hipMalloc((void**) &ac, sizeof(float) * lda * lda);
	hipMalloc((void**) &bc, sizeof(float) * ldb * ldb);
	hipMalloc((void**) &cc, sizeof(float) * ldc * ldc);



    hipMemcpy2D(ac, sizeof(float) * lda, a, sizeof(float) * lda, sizeof(float) * lda, lda, hipMemcpyHostToDevice);
	hipMemcpy2D(bc, sizeof(float) * ldb, b, sizeof(float) * ldb, sizeof(float) * ldb, ldb, hipMemcpyHostToDevice);
	int blocks = (ldc*ldc) / NUM_THREADS;
    // printf("blocks %d\n ",blocks);
    // Conv<<<blocks, NUM_THREADS>>>(ac, lda, bc, ldb, cc, ldc);
    Conv<<<blocks, NUM_THREADS,sizeof(float) * ldb*ldb>>>(ac, lda, bc, ldb, cc, ldc);

    //函式名稱<<<block 數目, thread 數目, shared memory 大小>>>(參數...);

    hipMemcpy2D(c, sizeof(float) * ldc, cc, sizeof(float) * ldc, sizeof(float) *ldc, ldc, hipMemcpyDeviceToHost);

	hipFree(ac);
	hipFree(bc);
	hipFree(cc);

	end = clock();
	return end-start;
}



bool InitCUDA()
{
	int count;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}

	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

int main(int argc, char* argv[]) {//name of input file and output file 
    FILE *fp = fopen(argv[1], "rb");
    FILE *outfile = fopen(argv[2], "wb"); 
    if(!fp) { 
        perror("無法讀取檔案"); 
        return EXIT_FAILURE; 
    } 
    char  *filesize = strtok(argv[1], ".");
    // printf("%s\n", filesize); //輸出www

    float *img, *kernel, *output;

	int kl = 3;
    int l = atoi(filesize);
    unsigned char ch; 

    img = (float*) malloc(sizeof(float) * l * l);
	kernel = (float*) malloc(sizeof(float) * kl * kl);
	output = (float*) malloc(sizeof(float) * (l-kl+1) * (l-kl+1));
    // gauss = (float*) malloc(sizeof(float) * 5 * 5);

    kernel[0] = 1;
	kernel[1] = 2;
	kernel[2] = 1;
	kernel[3] = 0;
	kernel[4] = 0;
	kernel[5] = 0;
	kernel[6] = -1;
	kernel[7] = -2;
	kernel[8] = -1;



    int wcount = 0; 
    while(!feof(fp)) 
    { 
        fread(&ch, sizeof(char), 1, fp);
        img[wcount] = int(ch);
        // printf(" %.0f ",img[wcount]);
        
        wcount++; 
        // if(wcount % 320==0) {  // 換行 
            // putchar('\n');
        // } 
    } 
    // putchar('\n');

    // for (int i=0;i<320;i++)
    // {
    //     printf(" %.0f ",img[i]);
    // }

    clock_t time = conv(img, l, kernel, kl, output, l-kl+1);


    if(!outfile) { 
            puts("檔案輸出失敗"); 
            return 1; 
        }

    for(int w=0;w<(l-kl+1) * (l-kl+1);w++)
    {
        ch = float(output[w]);
        // printf(" %03d ",int(ch));
        fwrite(&ch, sizeof(char), 1, outfile);
    }

	double sec = (double) time / CLOCKS_PER_SEC;
	printf("\nTime used: %.4lf   (%.2lf GFLOPS)\n", sec, 2.0 * l * l * l / (sec * 1E9));

    fclose(fp);
    fclose(outfile);

	free(img);
	free(kernel);
	free(output);
    
    return 0; 
} 