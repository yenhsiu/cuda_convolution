#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <hip/hip_runtime.h>

#define NUM_THREADS 256
#define BLOCK_SIZE	16


__global__ static void Gauss(const float* a, int lda, const float* b, int ldb, float* c, int ldc)
{
	extern __device__ float data[];
	const int tid = threadIdx.x;
	const int bid = blockIdx.x;
    const int idx = bid * blockDim.x + tid;
	const int row = idx / ldc;
	const int column = idx % ldc;
	int i,  j;
	
	for(i = 0; i < ldb*ldb; i ++){
		data[i] = b[i];
	}
	__syncthreads();
    // printf("bid %d \n",bid); 
    if(row < ldc && column < ldc) 
	{
		float t = 0;
		for (j = 0;j<ldb*ldb;j++)
		{
			t += a[ lda*row + column + lda*(j/ldc) + j%ldc ] * data[j];	     
		}
        // printf("t %.0f ",t);
        // printf("c %d \t",bid * blockDim.x + tid);
        if(t/256>255){c[bid * blockDim.x + tid]=255;}
        else if(t/256<0){ c[bid * blockDim.x + tid]=0;}
        else{ c[idx] = t/256;}
		// printf(" %.0f ",c[row * ldc + column]);
	}
}


clock_t gaussConv(const float* a, int lda, const float* b, int ldb, float* c, int ldc)
{
	float *ac, *bc, *cc;
	clock_t start, end;

	start = clock();

    hipMalloc((void**) &ac, sizeof(float) * lda * lda);
	hipMalloc((void**) &bc, sizeof(float) * ldb * ldb);
	hipMalloc((void**) &cc, sizeof(float) * ldc * ldc);



    hipMemcpy2D(ac, sizeof(float) * lda, a, sizeof(float) * lda, sizeof(float) * lda, lda, hipMemcpyHostToDevice);
	hipMemcpy2D(bc, sizeof(float) * ldb, b, sizeof(float) * ldb, sizeof(float) * ldb, ldb, hipMemcpyHostToDevice);
	int blocks = (ldc*ldc) / NUM_THREADS;
    // printf("blocks %d\n ",blocks);
    // Conv<<<blocks, NUM_THREADS>>>(ac, lda, bc, ldb, cc, ldc);
    Gauss<<<blocks, NUM_THREADS,sizeof(float) * ldb*ldb>>>(ac, lda, bc, ldb, cc, ldc);

    //函式名稱<<<block 數目, thread 數目, shared memory 大小>>>(參數...);

    hipMemcpy2D(c, sizeof(float) * ldc, cc, sizeof(float) * ldc, sizeof(float) *ldc, ldc, hipMemcpyDeviceToHost);

	hipFree(ac);
	hipFree(bc);
	hipFree(cc);

	end = clock();
	return end-start;
}



bool InitCUDA()
{
	int count;

	hipGetDeviceCount(&count);
	if(count == 0) {
		fprintf(stderr, "There is no device.\n");
		return false;
	}

	int i;
	for(i = 0; i < count; i++) {
		hipDeviceProp_t prop;
		if(hipGetDeviceProperties(&prop, i) == hipSuccess) {
			if(prop.major >= 1) {
				break;
			}
		}
	}

	if(i == count) {
		fprintf(stderr, "There is no device supporting CUDA 1.x.\n");
		return false;
	}

	hipSetDevice(i);

	return true;
}

int main(int argc, char* argv[]) {//name of input file and output file 
    FILE *fp = fopen(argv[1], "rb");
    FILE *outfile = fopen(argv[2], "wb"); 
    if(!fp) { 
        perror("無法讀取檔案"); 
        return EXIT_FAILURE; 
    } 
    char  *filesize = strtok(argv[1], ".");
    // printf("%s\n", filesize); //輸出www

    float *img, *gauss, *output;

	int kl = 5;
    int l = atoi(filesize);
    unsigned char ch; 

    img = (float*) malloc(sizeof(float) * l * l);
	gauss = (float*) malloc(sizeof(float) * kl * kl);
	output = (float*) malloc(sizeof(float) * (l-kl+1) * (l-kl+1));
    // gauss = (float*) malloc(sizeof(float) * 5 * 5);

	gauss[0] = 1;
    gauss[1] = 4;
    gauss[2] = 6;
    gauss[3] = 4;
    gauss[4] = 1;

    gauss[5] = 4;
    gauss[6] = 16;
    gauss[7] = 24;
    gauss[8] = 16;
    gauss[9] = 4;

    gauss[10] = 6;
    gauss[11] = 24;
    gauss[12] = 36;
    gauss[13] = 24;
    gauss[14] = 6;

    gauss[15] = 4;
    gauss[16] = 16;
    gauss[17] = 24;
    gauss[18] = 16;
    gauss[19] = 4;

    gauss[20] = 1;
    gauss[21] = 4;
    gauss[22] = 6;
    gauss[23] = 4;
    gauss[24] = 1;



    int wcount = 0; 
    while(!feof(fp)) 
    { 
        fread(&ch, sizeof(char), 1, fp);
        img[wcount] = int(ch);
        // printf(" %.0f ",img[wcount]);
        
        wcount++; 
        // if(wcount % 320==0) {  // 換行 
            // putchar('\n');
        // } 
    } 
    // putchar('\n');

    // for (int i=0;i<320;i++)
    // {
    //     printf(" %.0f ",img[i]);
    // }

    clock_t time = gaussConv(img, l, gauss, kl, output, l-kl+1);


    if(!outfile) { 
            puts("檔案輸出失敗"); 
            return 1; 
        }

    for(int w=0;w<(l-kl+1) * (l-kl+1);w++)
    {
        ch = float(output[w]);
        // printf(" %03d ",int(ch));
        fwrite(&ch, sizeof(char), 1, outfile);
    }

	double sec = (double) time / CLOCKS_PER_SEC;
	printf("\nTime used: %.4lf   (%.2lf GFLOPS)\n", sec, 2.0 * l * l * l / (sec * 1E9));

    fclose(fp);
    fclose(outfile);

	free(img);
	free(gauss);
	free(output);
    
    return 0; 
} 